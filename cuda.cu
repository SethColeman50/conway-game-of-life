#include "hip/hip_runtime.h"
// Compile with (on gpu session on cluster): nvcc -O3 -arch=sm_86 cuda.cu -o cuda
// Run with: ./cuda <input_file> <output_file> <num_steps>

#include "matrix.hpp"
#include "helpers.c"
#include <time.h>
#include <stdio.h>

#include <hip/hip_runtime.h>

/**
 * Macro to check if a CUDA call has an error, and if it does, report it and
 * exit the program.
 */
#define CHECK(call)                                                       \
{                                                                         \
   const hipError_t error = call;                                        \
   if (error != hipSuccess)                                              \
   {                                                                      \
      printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
      printf("code:%d, reason: %s\n", error, hipGetErrorString(error));  \
      exit(1);                                                            \
   }                                                                      \
}

__global__ void update_cell_kernel(size_t rows, size_t cols, double* input, double* output) {
    size_t col = blockIdx.x*blockDim.x + threadIdx.x;
    size_t row = blockIdx.y*blockDim.y + threadIdx.y;

    // check if we are out of bounds
    if (row >= rows || col >= cols) {
        return;
    }

    // checking if we are at an edge or corner
    bool is_top_boundary = row == 0;
    bool is_bottom_boundary = row == rows-1;
    bool is_left_boundary = col == 0;
    bool is_right_boundary = col == cols-1;

    // get the number of neighbors of the cell at (row, col) while making sure we don't go out of bounds
    double num_of_neighbors = (is_top_boundary ? 0 : input[(row-1) * cols + col]) +  /* top */               \
        (is_bottom_boundary ? 0 : input[(row+1) * cols + col]) + /* bottom */                                \
        (is_left_boundary ? 0 : input[row * cols + (col-1)]) + /* left */                                    \
        (is_right_boundary ? 0 : input[row * cols + (col+1)]) + /* right */                                  \
        ((is_top_boundary || is_left_boundary) ? 0 : input[(row-1) * cols + (col-1)]) + /* top left */         \
        ((is_top_boundary || is_right_boundary) ? 0 : input[(row-1) * cols + (col+1)]) + /* top right */       \
        ((is_bottom_boundary || is_left_boundary) ? 0 : input[(row+1) * cols + (col-1)]) + /* bottom left */   \
        ((is_bottom_boundary || is_right_boundary) ? 0 : input[(row+1) * cols + (col+1)]); /* bottom right */  \

    // get the value of the cell at (row, col)
    double value = input[row * cols + col];

    // conway's game of life rules
    if (value == 1) {
        if (num_of_neighbors <= 1 || num_of_neighbors >= 4) {
            output[row * cols + col] = 0;
        } else {
           output[row * cols + col] = 1;
        }
    } else {
        if (num_of_neighbors == 3) {
            output[row * cols + col] = 1;
        } else {
            output[row * cols + col] = 0;
        }
    }
}

int main(int argc, const char* argv[]) {
    // parse arguments
    Matrix<double> input = Matrix<double>::from_csv(argv[1]);
    const char* output_file = argv[2];
    size_t rows = input.rows;
    size_t cols = input.cols;
    size_t num_steps = atoi(argv[3]);
    
    // start the timer
    struct timespec start, end;
    clock_gettime(CLOCK_MONOTONIC, &start);

    Matrix<double> output = Matrix<double>(rows, cols).fill_zeros();

    // Allocate GPU memory
    double* d_input;
    double* d_output;
    CHECK(hipMalloc(&d_input, rows*cols*sizeof(double)));
    CHECK(hipMalloc(&d_output, rows*cols*sizeof(double)));

    // Copy input matrix to GPU
    CHECK(hipMemcpy(d_input, input.data, rows*cols*sizeof(double), hipMemcpyHostToDevice));

    // Calculate grid and block sizes
    dim3 block_size(16, 16);
    dim3 grid_size((rows+block_size.x-1)/block_size.x, (cols+block_size.y-1)/block_size.y);

    // // resets output matrix before starting loop
    // CHECK(hipMemset(d_output, output.data, rows * cols * sizeof(double));)

    // Launch kernel
    for (size_t t = 0; t < num_steps; t++) {

        // updates all cells in the matrix
        update_cell_kernel<<<grid_size, block_size>>>(rows, cols, d_input, d_output);
    
        // Check for any errors launching the kernel
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("Error launching update_cell_kernel: %s\n", hipGetErrorString(err));
            return -1;
        }

        // Wait for GPU to finish before accessing on host
        hipDeviceSynchronize();

        // Swap input and output pointers
        double* temp = d_input;
        d_input = d_output;
        d_output = temp;
    }

    // Copy output matrix from GPU
    // Note: d_input now points to the output matrix
    CHECK(hipMemcpy(output.data, d_input, rows*cols*sizeof(double), hipMemcpyDeviceToHost));

    // Free GPU memory
    CHECK(hipFree(d_input));
    CHECK(hipFree(d_output));

    // get the elapsed time
    clock_gettime(CLOCK_MONOTONIC, &end);
    double time = end.tv_sec-start.tv_sec+(end.tv_nsec-start.tv_nsec)/1000000000.0;
    printf("Time: %g secs\n", time);


    print_matrix(output);

    // save the output matrix
    output.to_csv(output_file);

    // reset the device
    CHECK(hipDeviceReset());

    return 0;
}